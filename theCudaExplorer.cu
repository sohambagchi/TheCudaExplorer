#include "hip/hip_runtime.h"
#include <cstdio>
#include <chrono>

#include "theCudaExplorer.cuh"

char randString[PADDING_LENGTH];

// creates a random string for the object padding
void initRandString(int paddingSize) {
    for (int i = 0; i < paddingSize; i++) {
        randString[i] = rand() % 26 + 'a';
    }
}

// creates the shuffling order for the objects
void shuffleList(int * localOrder, int count) {
    for (int i = 0; i < count; i++) {
        int j = rand() % count;
        int tmp = localOrder[i];
        localOrder[i] = localOrder[j];
        localOrder[j] = tmp;
    }
}

#define SAFE(x) if (0 != x) { printf("Error: %d @ Line %d\n", x, __LINE__); abort(); }

void printHelp() {
    printf("-n <int> : Number of Objects\n");
    printf("-o <string> : The order of operations\n");
    printf("-m <string> : The type of memory to use (DRAM or UM)\n");
    printf("-c <string> : Memory Order (acq, rel, acq_rel, acq_acq) for CPU Operations\n");
    printf("-g <string> : Memory Order (acq, rel, acq_rel, acq_acq) for CPU Operations\n");
    printf("-l <1, 1K, 10K, 100K, 1M, 10M, 100M> : The number of outer-loop iterations \n");
    printf("-t <string> : The type of Array being used (array or linkedlist)\n");
    printf("-w : Include if warmup required\n");

    printf("Example: ./theCudaExplorer -n 1024 -o \"PcCgg\" -m DRAM\n");
    printf("This runs the following Litmus Test using hipHostMalloc\n\n");
    printf("CPU st\n\t\tGPU ld\n\t\tGPU ld\n\n");
}

CEOperation * parseOperations(const char* operations) { 

    int total_operations = 0;

    for (int i = 0; i < strlen(operations); i++) {
        if (operations[i] == 'c' || operations[i] == 'g') total_operations++;
    }

    CEOperation * sequence = (CEOperation *) malloc(sizeof(CEOperation) * total_operations);

    CEAction mode;
    for (int i = 0, j = 0; i < strlen(operations), j < total_operations; i++) {
        
        switch (operations[i]) {
            case 'P':
                mode = CE_STORE;
                break;
            case 'C':
                mode = CE_LOAD;
                break;
            case 'g':
                sequence[j].total = total_operations;
                sequence[j].device = CE_GPU;
                sequence[j++].action = mode;
                break;
            case 'c':
                sequence[j].total = total_operations;
                sequence[j].device = CE_CPU;
                sequence[j++].action = mode;
                break;
            default:
                printf("Invalid Operation: %c\n", operations[i]);
                abort();
        }
    }
    return sequence;
}

void printSequence(CEOperation * sequence) {

    printf("Sequence\n======================\n");

    for (int i = 0; i < sequence[0].total; i++) {
        printf("%s %s\n", sequence[i].device == CE_CPU ? "CPU" : "\t\tGPU", sequence[i].action == CE_LOAD ? "ld" : "st");
    }

    printf("\n");

}

void printResults(CEOperation * sequence, int numCPUEvents, int numGPUEvents, int64_t * durations, unsigned int * loopDurations, float * milliseconds, int * count) {
    
    printf("Results\n======================\n");

    for (int i = 0, j = 0, k = 0; i < sequence[0].total; i++) {
        printf("%s %s\n", sequence[i].device == CE_CPU ? "CPU" : "\t\tGPU", sequence[i].action == CE_LOAD ? "ld" : "st");
        printf("%s    (%ld ns)\t[%ld ns]\n", sequence[i].device == CE_CPU ? "" : "\t\t", sequence[i].device == CE_CPU ? (durations[j] / *count) : (int64_t) (milliseconds[k] / *count), sequence[i].device == CE_CPU ? durations[j] : (int64_t) (milliseconds[k]));
        printf("%s    (%u cycles)\t[%u ns]\n", sequence[i].device == CE_CPU ? "" : "\t\t", sequence[i].device == CE_CPU ? 0 : (loopDurations[k] / *count), sequence[i].device == CE_CPU ? 0 : loopDurations[k]);
        if (sequence[i].device == CE_CPU) j++;
        else k++;
    }

}

int main(int argc, char* argv[]) {
    srand(time(NULL));

    if (argc < 2) {
        printHelp();
        return 0;
    }

    CEMemory memoryType;
    CEOperation * operationSequence;
    CEOrder gpuMemoryOrder;
    CEOrder cpuMemoryOrder;
    CECount outerLoopCount;
    CEObjectType objectType;
    int numObjects;

    bool warmup = false;
    
    int opt;
    while ((opt = getopt(argc, argv, "n:o:m:c:g:l:t:wh")) != -1) {
        switch (opt) {
            case 'n':
                numObjects = atoi(optarg);
                break;
            case 'o':
                operationSequence = parseOperations(optarg);
                break;
            case 'm':
                if (strcmp(optarg, "DRAM") == 0) {
                    memoryType = CE_DRAM;
                } else if (strcmp(optarg, "UM") == 0) {
                    memoryType = CE_UM;
                } else if (strcmp(optarg, "GDDR") == 0) {
                    memoryType = CE_GDDR;
                } else if (strcmp(optarg, "SYS") == 0) {
                    memoryType = CE_SYS;
                } else {
                    printf("Invalid Memory Type: %s\n", optarg);
                    abort();
                }
                break;
            case 'g':
                if (strcmp(optarg, "acq") == 0) {
                    gpuMemoryOrder = CE_ACQ;
                } else if (strcmp(optarg, "rel") == 0) {
                    gpuMemoryOrder = CE_REL;
                } else if (strcmp(optarg, "acq-rel") == 0) {
                    gpuMemoryOrder = CE_ACQ_REL;
                } else if (strcmp(optarg, "acq-acq") == 0) {
                    gpuMemoryOrder = CE_ACQ_ACQ;
                } else if (strcmp(optarg, "none") == 0) {
                    gpuMemoryOrder = CE_NONE;
                } else {
                    printf("Invalid Memory Order: %s\n", optarg);
                    abort();
                }
                break;
            case 'c': 
                if (strcmp(optarg, "acq") == 0) {
                    cpuMemoryOrder = CE_ACQ;
                } else if (strcmp(optarg, "rel") == 0) {
                    cpuMemoryOrder = CE_REL;
                } else if (strcmp(optarg, "acq-rel") == 0) {
                    cpuMemoryOrder = CE_ACQ;
                } else if (strcmp(optarg, "acq-acq") == 0) {
                    cpuMemoryOrder = CE_ACQ;
                } else if (strcmp(optarg, "none") == 0) {
                    cpuMemoryOrder = CE_NONE;
                } else {
                    printf("Invalid Memory Order: %s\n", optarg);
                    abort();
                }
                break;
            case 'l':
                if (strcmp(optarg, "1K") == 0) {
                    outerLoopCount = CE_1K;
                } else if (strcmp(optarg, "10K") == 0) {
                    outerLoopCount = CE_10K;
                } else if (strcmp(optarg, "100K") == 0) {
                    outerLoopCount = CE_100K;
                } else if (strcmp(optarg, "1M") == 0) {
                    outerLoopCount = CE_1M;
                } else if (strcmp(optarg, "10M") == 0) {
                    outerLoopCount = CE_10M;
                } else if (strcmp(optarg, "100M") == 0) {
                    outerLoopCount = CE_100M;
                } else if (strcmp(optarg, "1B") == 0) {
                    outerLoopCount = CE_1B;
                } else if (strcmp(optarg, "1") == 0) {
                    outerLoopCount = CE_BASE;
                } else {
                    printf("Invalid Loop Count: %s\n", optarg);
                    abort();
                }
                break;
            case 'w':
                warmup = true;
                break;
            case 't':
                if (strcmp(optarg, "array") == 0) {
                    objectType = CE_ARRAY;
                } else if (strcmp(optarg, "linkedlist") == 0) {
                    // printf("Using LinkedList\n");
                    objectType = CE_LINKEDLIST;
                } else {
                    printf("Invalid Array Type: %s\n", optarg);
                    abort();
                }
                break;
            case 'h':
                printHelp();
                return 0;
            default:
                printHelp();
                return 0;
        }
    }

    int numCPUEvents = 0, numGPUEvents = 0;

    int GPUConsumeFirst = -1;

    // identify CPU and GPU events separately for timers
    for (int i = 0; i < operationSequence[0].total; i++) {
        // printf("%s %s\n", operationSequence[i].device == CE_CPU ? "CPU" : "\t\tGPU", operationSequence[i].action == CE_LOAD ? "ld" : "st");
        if (operationSequence[i].device == CE_CPU) {
            numCPUEvents++;
        } else {
            numGPUEvents++;
            if (GPUConsumeFirst == -1 && operationSequence[i].action == CE_LOAD) {
                GPUConsumeFirst = 1;
            } else if (GPUConsumeFirst == -1 && operationSequence[i].action == CE_STORE) {
                GPUConsumeFirst = 0;
            }
        }
    }

    // CPU Timers
    std::chrono::high_resolution_clock::time_point begin[numCPUEvents], end[numCPUEvents];
    int64_t durations[numCPUEvents];

    // CUDA Timers
    hipEvent_t start[numGPUEvents], stop[numGPUEvents];
    float milliseconds[numGPUEvents];

    // Internal clock64() timers
    unsigned int * beforeLoop[numGPUEvents];
    unsigned int * afterLoop[numGPUEvents];
    unsigned int * localBeforeLoop[numGPUEvents];
    unsigned int * localAfterLoop[numGPUEvents];
    unsigned int loopDuration[numGPUEvents];

    for (int i = 0; i < numGPUEvents; i++) {
        SAFE(hipEventCreate(&start[i]));
        SAFE(hipEventCreate(&stop[i]));
        SAFE(hipMalloc(&beforeLoop[i], sizeof(unsigned int)));
        SAFE(hipMalloc(&afterLoop[i], sizeof(unsigned int)));

        localBeforeLoop[i] = (unsigned int *) calloc(1, sizeof(unsigned int));

        if (localBeforeLoop[i] == NULL) {
            printf("Failed to allocate memory for localBeforeLoop\n");
            abort();
        }

        localAfterLoop[i] = (unsigned int *) calloc(1, sizeof(unsigned int));

        if (localAfterLoop[i] == NULL) {
            printf("Failed to allocate memory for localAfterLoop\n");
            abort();
        }
    }

    cuda::atomic<int>* flag;
    struct LargeLinkedObject * largeObjectList;
    int * largeObjectListConsumer;
    int * localConsumer;
    int * largeObjectListOrder;
    int * localOrder;
    int * count;

    SAFE(hipHostMalloc(&count, sizeof(int)));

    *count = numObjects;

    printf("Size of Object: %.2f MB, %.2f KB\n", sizeof(struct LargeLinkedObject) / (1024.0 * 1024.0), sizeof(struct LargeLinkedObject) / 1024.0);
    printf("Number of Objects: %d\n", *count);
    printf("CPU Events Timed: %d\t GPU Events Timed: %d\n", numCPUEvents, numGPUEvents);

    SAFE(hipHostMalloc(&flag, sizeof(cuda::atomic<int>)));
    SAFE(hipHostMalloc(&largeObjectListConsumer, sizeof(int) * *count));
    SAFE(hipHostMalloc(&localConsumer, sizeof(int) * *count));
    SAFE(hipHostMalloc(&largeObjectListOrder, sizeof(int) * *count));
    SAFE(hipHostMalloc(&localOrder, sizeof(int) * *count));

    if (memoryType == CE_DRAM) {
        SAFE(hipHostMalloc(&largeObjectList, sizeof(struct LargeLinkedObject) * *count));
    } else if (memoryType == CE_UM) {
        SAFE(hipMallocManaged(&largeObjectList, sizeof(struct LargeLinkedObject) * *count));
    } else if (memoryType == CE_GDDR) {
        SAFE(hipMalloc(&largeObjectList, sizeof(struct LargeLinkedObject) * *count));
    } else {
        largeObjectList = (struct LargeLinkedObject*) malloc(sizeof(struct LargeLinkedObject) * *count);
    }

    // allocate the ordering array in both DRAM and GDDR
    for (int i = 0; i < (*count); i++) {
        localOrder[i] = i;
    }

    shuffleList(localOrder, *count);
    SAFE(hipMemcpy(largeObjectListOrder, localOrder, sizeof(int) * *count, hipMemcpyHostToDevice));

    printf("\nUsing %s for Objects\n", memoryType == CE_DRAM ? "hipHostMalloc" : memoryType == CE_UM ? "hipMallocManaged" : memoryType == CE_GDDR ? "hipMalloc" : "malloc");
    printf("Per-Iteration Loads: %s\n\n", 
    gpuMemoryOrder == CE_ACQ ? "acquire" : gpuMemoryOrder == CE_REL ? "release" : gpuMemoryOrder == CE_ACQ_ACQ ? "acq/acq" : gpuMemoryOrder == CE_ACQ_REL ? "acq/rel" : "non-atomic");


    //randomly pad all the objects
    if (memoryType == CE_GDDR) {
        struct LargeLinkedObject * localCopy = (struct LargeLinkedObject *) malloc(sizeof(struct LargeLinkedObject) * *count);

        for (int i = 0; i < (*count); i++) {
            initRandString((sizeof(LargeLinkedObject) - sizeof(int)) / (2 * sizeof(char)));

            strcpy(localCopy[localOrder[i]].padding1, randString);
            // initRandString((sizeof(LargeLinkedObject) - sizeof(int)) / (2 * sizeof(char)));
            strcpy(localCopy[localOrder[i]].padding2, randString);

            localCopy[localOrder[i]].data_na = localOrder[(i + 1) % *count];
            localCopy[localOrder[i]].data.store(localOrder[(i + 1) % *count]);
        }

        SAFE(hipMemcpy(largeObjectList, localCopy, sizeof(struct LargeLinkedObject) * *count, hipMemcpyHostToDevice));

    } else {
        for (int i = 0; i < (*count); i++) {
            initRandString((sizeof(LargeLinkedObject) - sizeof(int)) / (2 * sizeof(char)));

            strcpy(largeObjectList[localOrder[i]].padding1, randString);
            // initRandString((sizeof(LargeLinkedObject) - sizeof(int)) / (2 * sizeof(char)));
            strcpy(largeObjectList[localOrder[i]].padding2, randString);

            largeObjectList[localOrder[i]].data_na = localOrder[(i + 1) % *count];
            largeObjectList[localOrder[i]].data.store(localOrder[(i + 1) % *count]);
        }
    }

    printSequence(operationSequence);

    int CPUEventCount = 0;
    int GPUEventCount = 0;

    if (warmup && GPUConsumeFirst == 1) {
        printf("Warming up GPU\n");
        switch (gpuMemoryOrder) {
            case CE_ACQ:
                if (objectType == CE_ARRAY){
                    GPUListConsumer_acq<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, largeObjectListOrder, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);}
                else
                    GPULinkedListConsumer_acq<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                break;
            case CE_REL:
                if (objectType == CE_ARRAY)
                    GPUListConsumer_rel<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, largeObjectListOrder, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                else 
                    GPULinkedListConsumer_rel<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                break;
            case CE_ACQ_ACQ:
                if (objectType == CE_ARRAY)
                    GPUListConsumer_acq_acq<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, largeObjectListOrder, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                else
                    GPULinkedListConsumer_acq_acq<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                break;
            case CE_ACQ_REL:
                if (objectType == CE_ARRAY)
                    GPUListConsumer_acq_rel<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, largeObjectListOrder, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                else
                    GPULinkedListConsumer_acq_rel<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                break;
            case CE_NONE:
                if (objectType == CE_ARRAY)
                    GPUListConsumer<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, largeObjectListOrder, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                else
                    GPULinkedListConsumer<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                break;
        }
        hipDeviceSynchronize();
    } else {
        printf("Skipping GPU Warmup\n");
    }

    // do all the operations in the order specified
    for (int i = 0; i < operationSequence[0].total; i++) {
        if (operationSequence[i].device == CE_CPU) {
            begin[CPUEventCount] = std::chrono::high_resolution_clock::now();
            switch (operationSequence[i].action) {
                case CE_LOAD:
                    switch (cpuMemoryOrder) {
                        case CE_ACQ:
                            CPUListConsumer_acq(flag, largeObjectList, localConsumer, localOrder, count);
                            break;
                        case CE_REL:
                            CPUListConsumer_rel(flag, largeObjectList, localConsumer, localOrder, count);
                            break;
                        case CE_NONE:
                            if (objectType == CE_ARRAY)
                                CPUListConsumer(flag, largeObjectList, localConsumer, localOrder, count);
                            else {
                                if (outerLoopCount == CE_1K) 
                                    CPULinkedListConsumer_1K(flag, largeObjectList, localConsumer, count);
                                else 
                                    CPULinkedListConsumer(flag, largeObjectList, localConsumer, count);
                            }
                            break;
                    }
                    break;
                case CE_STORE:
                    switch (cpuMemoryOrder) {
                        case CE_NONE:
                            if (objectType == CE_ARRAY)
                                CPUListProducer(flag, largeObjectList, localOrder, count);
                            else
                                CPULinkedListProducer(flag, largeObjectList, localOrder, count);
                            break;
                        case CE_ACQ:
                        case CE_REL:
                            CPUListProducer_rel(flag, largeObjectList, localOrder, count);
                            break;
                    }
                    break;
            }
            end[CPUEventCount++] = std::chrono::high_resolution_clock::now();
        } else {
            hipEventRecord(start[GPUEventCount]);
            switch (operationSequence[i].action) {
                case CE_LOAD:
                    switch (gpuMemoryOrder) {
                        case CE_ACQ:
                            switch (outerLoopCount) {
                                case CE_1K:
                                    if (objectType == CE_ARRAY)
                                        GPUListConsumer_acq_1K<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, largeObjectListOrder, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    else
                                        GPULinkedListConsumer_acq_1K<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    break;
                                case CE_10K:
                                    if (objectType == CE_ARRAY)
                                        GPUListConsumer_acq_10K<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, largeObjectListOrder, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    else
                                        GPULinkedListConsumer_acq_10K<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    break;
                                case CE_100K:
                                    if (objectType == CE_ARRAY)
                                        GPUListConsumer_acq_100K<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, largeObjectListOrder, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    else
                                        GPULinkedListConsumer_acq_100K<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    break;
                                case CE_1M:
                                    if (objectType == CE_ARRAY)
                                        GPUListConsumer_acq_1M<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, largeObjectListOrder, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    else
                                        GPULinkedListConsumer_acq_1M<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    break;
                                case CE_10M:
                                    if (objectType == CE_ARRAY)
                                        GPUListConsumer_acq_10M<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, largeObjectListOrder, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    else
                                        GPULinkedListConsumer_acq_10M<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    break;
                                case CE_100M:
                                    if (objectType == CE_ARRAY)
                                        GPUListConsumer_acq_100M<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, largeObjectListOrder, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    else
                                        GPULinkedListConsumer_acq_100M<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    break;
                                case CE_1B:
                                    GPUListConsumer_acq_1B<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, largeObjectListOrder, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    break;
                                case CE_BASE:
                                    if (objectType == CE_ARRAY)
                                        GPUListConsumer_acq<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, largeObjectListOrder, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    else
                                        GPULinkedListConsumer_acq<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    break;
                            }
                            break;
                        case CE_REL:
                            switch (outerLoopCount) {
                                case CE_1K:
                                    if (objectType == CE_ARRAY)
                                        GPUListConsumer_rel_1K<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, largeObjectListOrder, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    else
                                        GPULinkedListConsumer_rel_1K<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    break;
                                case CE_10K:
                                    if (objectType == CE_ARRAY)
                                        GPUListConsumer_rel_10K<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, largeObjectListOrder, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    else
                                        GPULinkedListConsumer_rel_10K<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    break;
                                case CE_100K:
                                    if (objectType == CE_ARRAY)
                                        GPUListConsumer_rel_100K<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, largeObjectListOrder, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    else
                                        GPULinkedListConsumer_rel_100K<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    break;
                                case CE_1M:
                                    if (objectType == CE_ARRAY)
                                        GPUListConsumer_rel_1M<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, largeObjectListOrder, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    else
                                        GPULinkedListConsumer_rel_1M<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    break;
                                case CE_10M:
                                    if (objectType == CE_ARRAY)
                                        GPUListConsumer_rel_10M<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, largeObjectListOrder, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    else
                                        GPULinkedListConsumer_rel_10M<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    break;
                                case CE_100M:
                                    if (objectType == CE_ARRAY)
                                        GPUListConsumer_rel_100M<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, largeObjectListOrder, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    else
                                        GPULinkedListConsumer_rel_100M<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    break;
                                case CE_1B:
                                    GPUListConsumer_rel_1B<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, largeObjectListOrder, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    break;
                                case CE_BASE:
                                    if (objectType == CE_ARRAY)
                                        GPUListConsumer_rel<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, largeObjectListOrder, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    else
                                        GPULinkedListConsumer_rel<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    break;
                            }
                            break;
                        case CE_ACQ_ACQ:
                            switch (outerLoopCount) {
                                case CE_1K:
                                    if (objectType == CE_ARRAY)
                                        GPUListConsumer_acq_acq_1K<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, largeObjectListOrder, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    else
                                        GPULinkedListConsumer_acq_acq_1K<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    break;
                                case CE_10K:
                                    if (objectType == CE_ARRAY)
                                        GPUListConsumer_acq_acq_10K<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, largeObjectListOrder, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    else
                                        GPULinkedListConsumer_acq_acq_10K<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    break;
                                case CE_100K:
                                    if (objectType == CE_ARRAY)
                                        GPUListConsumer_acq_acq_100K<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, largeObjectListOrder, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    else
                                        GPULinkedListConsumer_acq_acq_100K<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    break;
                                case CE_1M:
                                    if (objectType == CE_ARRAY)
                                        GPUListConsumer_acq_acq_1M<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, largeObjectListOrder, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    else
                                        GPULinkedListConsumer_acq_acq_1M<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    break;
                                case CE_10M:
                                    if (objectType == CE_ARRAY)
                                        GPUListConsumer_acq_acq_10M<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, largeObjectListOrder, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    else
                                        GPULinkedListConsumer_acq_acq_10M<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    break;
                                case CE_100M:
                                    if (objectType == CE_ARRAY)
                                        GPUListConsumer_acq_acq_100M<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, largeObjectListOrder, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    else
                                        GPULinkedListConsumer_acq_acq_100M<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    break;
                                case CE_1B:
                                    GPUListConsumer_acq_acq_1B<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, largeObjectListOrder, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    break;
                                case CE_BASE:
                                    if (objectType == CE_ARRAY)
                                        GPUListConsumer_acq_acq<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, largeObjectListOrder, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    else
                                        GPULinkedListConsumer_acq_acq<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    break;
                            }
                            break;
                        case CE_ACQ_REL:
                            switch (outerLoopCount) {
                                case CE_1K:
                                    if (objectType == CE_ARRAY)
                                        GPUListConsumer_acq_rel_1K<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, largeObjectListOrder, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    else
                                        GPULinkedListConsumer_acq_rel_1K<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    break;
                                case CE_10K:
                                    if (objectType == CE_ARRAY)
                                        GPUListConsumer_acq_rel_10K<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, largeObjectListOrder, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    else
                                        GPULinkedListConsumer_acq_rel_10K<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    break;
                                case CE_100K:
                                    if (objectType == CE_ARRAY)
                                        GPUListConsumer_acq_rel_100K<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, largeObjectListOrder, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    else
                                        GPULinkedListConsumer_acq_rel_100K<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    break;
                                case CE_1M:
                                    if (objectType == CE_ARRAY)
                                        GPUListConsumer_acq_rel_1M<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, largeObjectListOrder, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    else
                                        GPULinkedListConsumer_acq_rel_1M<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    break;
                                case CE_10M:
                                    if (objectType == CE_ARRAY)
                                        GPUListConsumer_acq_rel_10M<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, largeObjectListOrder, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    else
                                        GPULinkedListConsumer_acq_rel_10M<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    break;
                                case CE_100M:
                                    if (objectType == CE_ARRAY)
                                        GPUListConsumer_acq_rel_100M<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, largeObjectListOrder, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    else
                                        GPULinkedListConsumer_acq_rel_100M<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    break;
                                case CE_1B:
                                    GPUListConsumer_acq_rel_1B<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, largeObjectListOrder, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    break;
                                case CE_BASE:
                                    if (objectType == CE_ARRAY)
                                        GPUListConsumer_acq_rel<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, largeObjectListOrder, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    else
                                        GPULinkedListConsumer_acq_rel<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    break;
                            }
                            break;
                        case CE_NONE:
                            switch (outerLoopCount) {
                                case CE_1K:
                                    if (objectType == CE_ARRAY)
                                        GPUListConsumer_1K<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, largeObjectListOrder, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    else
                                        GPULinkedListConsumer_1K<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    break;
                                case CE_10K:
                                    if (objectType == CE_ARRAY)
                                        GPUListConsumer_10K<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, largeObjectListOrder, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    else
                                        GPULinkedListConsumer_10K<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    break;
                                case CE_100K:
                                    if (objectType == CE_ARRAY)
                                        GPUListConsumer_100K<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, largeObjectListOrder, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    else
                                        GPULinkedListConsumer_100K<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    break;
                                case CE_1M:
                                    if (objectType == CE_ARRAY)
                                        GPUListConsumer_1M<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, largeObjectListOrder, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    else
                                        GPULinkedListConsumer_1M<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    break;
                                case CE_10M:
                                    if (objectType == CE_ARRAY)
                                        GPUListConsumer_10M<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, largeObjectListOrder, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    else
                                        GPULinkedListConsumer_10M<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    break;
                                case CE_100M:
                                    if (objectType == CE_ARRAY)
                                        GPUListConsumer_100M<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, largeObjectListOrder, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    else
                                        GPULinkedListConsumer_100M<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    break;
                                case CE_1B:
                                    GPUListConsumer_1B<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, largeObjectListOrder, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    break;
                                case CE_BASE:
                                    if (objectType == CE_ARRAY)
                                        GPUListConsumer<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, largeObjectListOrder, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    else
                                        GPULinkedListConsumer<<<1,1>>>(flag, largeObjectList, largeObjectList, largeObjectListConsumer, count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                                    break;
                            }
                            break;
                    }
                    // printf("GPU Consumer %d %d %p %u %p %u\n", objectType, GPUEventCount, localAfterLoop[GPUEventCount], *localAfterLoop[GPUEventCount], localBeforeLoop[GPUEventCount], *localBeforeLoop[GPUEventCount]);
                    // SAFE(hipMemcpy(localAfterLoop[GPUEventCount], afterLoop[GPUEventCount], sizeof(unsigned int), hipMemcpyDeviceToHost));
                    // SAFE(hipMemcpy(localBeforeLoop[GPUEventCount], beforeLoop[GPUEventCount], sizeof(unsigned int), hipMemcpyDeviceToHost));
                    // printf("GPU Consumer %d %d %p %u %p %u\n", objectType, GPUEventCount, localAfterLoop[GPUEventCount], *localAfterLoop[GPUEventCount], localBeforeLoop[GPUEventCount], *localBeforeLoop[GPUEventCount]);
                    break;
                case CE_STORE:
                    switch (gpuMemoryOrder) {
                        case CE_NONE:
                            EmptyKernel<<<1,1>>>(count, beforeLoop[GPUEventCount], afterLoop[GPUEventCount]);
                            // GPUListProducer<<<1,1>>>(flag, largeObjectList, largeObjectListOrder, count);
                            break;
                        case CE_ACQ:
                        case CE_REL:
                            GPUListProducer_rel<<<1,1>>>(flag, largeObjectList, largeObjectListOrder, count);
                            break;
                    }
                    break;
            }
            hipEventRecord(stop[GPUEventCount]);

            // synchronize GPU executation after every operation
            hipEventSynchronize(stop[GPUEventCount]);
            GPUEventCount++;
        }
    }

    hipDeviceSynchronize();

    for (int i = 0; i < numGPUEvents; i++) {
        milliseconds[i] = 0;
        hipEventElapsedTime(&milliseconds[i], start[i], stop[i]);
        milliseconds[i] *= 1e6;
        loopDuration[i] = localAfterLoop[i] - localBeforeLoop[i];
    }

    for (int i = 0; i < numCPUEvents; i++) {
        durations[i] = std::chrono::duration_cast<std::chrono::nanoseconds>(end[i] - begin[i]).count();
    }

    printResults(operationSequence, numCPUEvents, numGPUEvents, durations, loopDuration, milliseconds, count);

    printf("\n----------------------\n\n\n");

    return 0;
}
