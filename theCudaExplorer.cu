#include <cstdio>
#include <chrono>

#include "theCudaExplorer.cuh"

char randString[3556156];

// creates a random string for the object padding
void initRandString(int paddingSize) {
    for (int i = 0; i < paddingSize; i++) {
        randString[i] = rand() % 26 + 'a';
    }
}

// creates the shuffling order for the objects
void shuffleList(int ** localOrder, int count) {
    for (int i = 0; i < count; i++) {
        int j = rand() % count;
        int *tmp = localOrder[i];
        localOrder[i] = localOrder[j];
        localOrder[j] = tmp;
    }
}

#define SAFE(x) if (0 != x) { abort(); }

void printHelp() {
    printf("-n <int> : Number of Objects\n");
    printf("-o <string> : The order of operations\n");
    printf("-m <string> : The type of memory to use (DRAM or UM)\n\n");

    printf("Example: ./theCudaExplorer -n 1024 -o \"PcCgg\" -m DRAM\n");
    printf("This runs the following Litmus Test using hipHostMalloc\n\n");
    printf("CPU st\n\t\tGPU ld\n\t\tGPU ld\n\n");
}

CEOperation * parseOperations(const char* operations) { 

    int total_operations = 0;

    for (int i = 0; i < strlen(operations); i++) {
        if (operations[i] == 'c' || operations[i] == 'g') total_operations++;
    }

    CEOperation * sequence = (CEOperation *) malloc(sizeof(CEOperation) * total_operations);

    CEAction mode;
    for (int i = 0, j = 0; i < strlen(operations), j < total_operations; i++) {
        
        switch (operations[i]) {
            case 'P':
                mode = CE_STORE;
                break;
            case 'C':
                mode = CE_LOAD;
                break;
            case 'g':
                sequence[j].total = total_operations;
                sequence[j].device = CE_GPU;
                sequence[j++].action = mode;
                break;
            case 'c':
                sequence[j].total = total_operations;
                sequence[j].device = CE_CPU;
                sequence[j++].action = mode;
                break;
            default:
                printf("Invalid Operation: %c\n", operations[i]);
                abort();
        }
    }
    return sequence;
}

void printSequence(CEOperation * sequence) {

    printf("Sequence\n======================\n");

    for (int i = 0; i < sequence[0].total; i++) {
        printf("%s %s\n", sequence[i].device == CE_CPU ? "CPU" : "\t\tGPU", sequence[i].action == CE_LOAD ? "ld" : "st");
    }

    printf("\n");

}

void printResults(CEOperation * sequence, int numCPUEvents, int numGPUEvents, int64_t * durations, float * milliseconds, int * count) {
    
    printf("Results\n======================\n");

    for (int i = 0, j = 0, k = 0; i < sequence[0].total; i++) {
        printf("%s %s\n", sequence[i].device == CE_CPU ? "CPU" : "\t\tGPU", sequence[i].action == CE_LOAD ? "ld" : "st");
        printf("%s    (%ld ns)\n", sequence[i].device == CE_CPU ? "" : "\t\t", sequence[i].device == CE_CPU ? durations[j++] / *count : (int64_t) (milliseconds[k++] / *count));
    }

}

int main(int argc, char* argv[]) {
    srand(time(NULL));

    if (argc < 2) {
        printHelp();
        return 0;
    }

    CEMemory memoryType;
    CEOperation * operationSequence;
    int numObjects;
    
        int opt;
    while ((opt = getopt(argc, argv, "n:o:m:h")) != -1) {
        switch (opt) {
            case 'n':
                numObjects = atoi(optarg);
                break;
            case 'o':
                operationSequence = parseOperations(optarg);
                break;
            case 'm':
                if (strcmp(optarg, "DRAM") == 0) {
                    memoryType = CE_DRAM;
                } else if (strcmp(optarg, "UM") == 0) {
                    memoryType = CE_UM;
                } else {
                    printf("Invalid Memory Type: %s\n", optarg);
                    abort();
                }
                break;
            case 'h':
                printHelp();
                return 0;
            default:
                printHelp();
                return 0;
        }
    }

    int numCPUEvents = 0, numGPUEvents = 0;

    // identify CPU and GPU events separately for timers
    for (int i = 0; i < operationSequence[0].total; i++) {
        // printf("%s %s\n", operationSequence[i].device == CE_CPU ? "CPU" : "\t\tGPU", operationSequence[i].action == CE_LOAD ? "ld" : "st");
        if (operationSequence[i].device == CE_CPU) {
            numCPUEvents++;
        } else {
            numGPUEvents++;
        }
    }

    // CPU Timers
    std::chrono::high_resolution_clock::time_point begin[numCPUEvents], end[numCPUEvents];
    int64_t durations[numCPUEvents];

    // CUDA Timers
    hipEvent_t start[numGPUEvents], stop[numGPUEvents];
    float milliseconds[numGPUEvents];

    for (int i = 0; i < numGPUEvents; i++) {
        SAFE(hipEventCreate(&start[i]));
        SAFE(hipEventCreate(&stop[i]));
    }

    cuda::atomic<int>* flag;
    struct LargeObject ** largeObjectList;
    int ** largeObjectListConsumer;
    int ** localConsumer;
    int ** largeObjectListOrder;
    int ** localOrder;
    int * count;

    SAFE(hipHostMalloc(&count, sizeof(int)));

    *count = numObjects;

    printf("Size of Object: %.2f MB\n", sizeof(struct LargeObject) / (1024.0 * 1024.0));
    printf("Number of Objects: %d\n", *count);
    printf("CPU Events Timed: %d\t GPU Events Timed: %d\n", numCPUEvents, numGPUEvents);

    SAFE(hipHostMalloc(&flag, sizeof(cuda::atomic<int>)));
    SAFE(hipHostMalloc(&largeObjectList, sizeof(struct LargeObject*) * *count));
    SAFE(hipHostMalloc(&largeObjectListConsumer, sizeof(int*) * *count));
    SAFE(hipHostMalloc(&localConsumer, sizeof(int*) * *count));
    SAFE(hipHostMalloc(&largeObjectListOrder, sizeof(int*) * *count));
    SAFE(hipHostMalloc(&localOrder, sizeof(int*) * *count));

    // allocate the ordering array in both DRAM and GDDR
    for (int i = 0; i < (*count); i++) {
        SAFE(hipMalloc(&largeObjectListOrder[i], sizeof(int)));
        SAFE(hipHostMalloc(&localOrder[i], sizeof(int)));
        *localOrder[i] = i;
    }

    shuffleList(localOrder, *count);

    for (int i = 0; i < (*count); i++) {

        // Allocate the data objects according to arguments
        if (memoryType == CE_DRAM) {
            SAFE(hipHostMalloc(&largeObjectList[*localOrder[i]], sizeof(struct LargeObject)));
        } else {
            SAFE(hipMallocManaged(&largeObjectList[*localOrder[i]], sizeof(struct LargeObject)));
        }

        // Separate Consumer Lists for CPU and GPU, to mitigate remote store latency
        SAFE(hipMalloc(&largeObjectListConsumer[i], sizeof(int)));
        SAFE(hipHostMalloc(&localConsumer[i], sizeof(int)));

        // Copy the locally shuffled order to the device
        SAFE(hipMemcpy(largeObjectListOrder[i], localOrder[i], sizeof(int), hipMemcpyHostToDevice));
    }

    printf("\nUsing %s for Objects\n\n", memoryType == CE_DRAM ? "hipHostMalloc" : "hipMallocManaged");

    printSequence(operationSequence);

    // randomly pad all the objects
    for (int i = 0; i < (*count); i++) {
        initRandString((sizeof(LargeObject) - sizeof(int)) / (2 * sizeof(char)));
        strcpy((largeObjectList[*localOrder[i]])->padding1, randString);
        (largeObjectList[*localOrder[i]])->data = i;
        strcpy((largeObjectList[*localOrder[i]])->padding2, randString);
    }

    int CPUEventCount = 0;
    int GPUEventCount = 0;

    // do all the operations in the order specified
    for (int i = 0; i < operationSequence[0].total; i++) {
        if (operationSequence[i].device == CE_CPU) {
            begin[CPUEventCount] = std::chrono::high_resolution_clock::now();
            switch (operationSequence[i].action) {
                case CE_LOAD:
                    CPUListConsumer(flag, largeObjectList, localConsumer, localOrder, count);
                    break;
                case CE_STORE:
                    CPUListProducer(flag, largeObjectList, localOrder, count);
                    break;
            }
            end[CPUEventCount++] = std::chrono::high_resolution_clock::now();
        } else {
            hipEventRecord(start[GPUEventCount]);
            switch (operationSequence[i].action) {
                case CE_LOAD:
                    GPUListConsumer<<<1,1>>>(flag, largeObjectList, largeObjectListConsumer, largeObjectListOrder, count);
                    break;
                case CE_STORE:
                    GPUListProducer<<<1,1>>>(flag, largeObjectList, largeObjectListOrder, count);
                    break;
            }
            hipEventRecord(stop[GPUEventCount]);

            // synchronize GPU executation after every operation
            hipEventSynchronize(stop[GPUEventCount]);
            GPUEventCount++;
        }
    }

    hipDeviceSynchronize();

    for (int i = 0; i < numGPUEvents; i++) {
        milliseconds[i] = 0;
        hipEventElapsedTime(&milliseconds[i], start[i], stop[i]);
        milliseconds[i] *= 1e6;
    }

    for (int i = 0; i < numCPUEvents; i++) {
        durations[i] = std::chrono::duration_cast<std::chrono::nanoseconds>(end[i] - begin[i]).count();
    }

    printResults(operationSequence, numCPUEvents, numGPUEvents, durations, milliseconds, count);

    return 0;


    // LABEL:MH40
    // struct LargeObject ** remoteObjectList;
    
    // SAFE(hipHostMalloc(&remoteObjectList, sizeof(struct LargeObject*) * *count));
      
    // for (int i = 0; i < (*count); i++) {
    //     SAFE(hipMalloc(&remoteObjectList[*localOrder[i]], sizeof(struct LargeObject)));
    // }

    // SAFE(hipMemcpy(remoteObjectList[*localOrder[i]], largeObjectList[*localOrder[i]], sizeof(struct LargeObject), hipMemcpyHostToDevice));

}